#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16

// kernel to unroll input matrix to prepare it for matrix multiplication
__global__ void input_matrix_unroll(const float *input, float *input_unroll, const int B, const int C, const int H, const int W, const int K, const int S)
{
#define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
  int H_out = (H - K) / S + 1;
  int W_out = (W - K) / S + 1;

  int unrolled_W = H_out * W_out;
  int unrolled_H = C * K * K;

  int w_unroll = blockIdx.x * blockDim.x + threadIdx.x;
  int h_unroll = blockIdx.y * blockDim.y + threadIdx.y;

  int b = blockIdx.z;
  int c = h_unroll / (K*K);

  int h = w_unroll / W_out;
  int w = w_unroll % W_out;

  int transposed_h = h_unroll - c * K * K;

  int p = transposed_h / K;
  int q = transposed_h % K;

  int input_h = h * S;
  int input_w = w * S;

  if (h < H_out && w < W_out && c < C && b < B && h_unroll < unrolled_H && w_unroll < unrolled_W && p < K && q < K && input_h + p < H && input_w + q < W)
  {
    input_unroll[b * (unrolled_H * unrolled_W) + h_unroll * unrolled_W + w_unroll] = in_4d(b, c, input_h + p, input_w + q);
  }
}

// kernel to perform shared memory matrix multiplication
__global__ void matrixMultiplyShared(const float *A, const float *B, float *C,
                                     int numARows, int numAColumns,
                                     int numBRows, int numBColumns,
                                     int numCRows, int numCColumns)
{
  //@@ Insert code to implement matrix multiplication here
  //@@ You have to use shared memory for this MP
  __shared__ float subTileA[TILE_WIDTH][TILE_WIDTH];
  __shared__ float subTileB[TILE_WIDTH][TILE_WIDTH];
  int bx = blockIdx.x;
  int by = blockIdx.y;
  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int b = blockIdx.z;
  // Identify the row and column of the C element to work on
  int Row = by * TILE_WIDTH + ty;
  int Col = bx * TILE_WIDTH + tx;
  float Pvalue = 0;
  // Loop over the A and B tiles required to compute the C element
  for (int q = 0; q < ((numAColumns + TILE_WIDTH - 1) / TILE_WIDTH); ++q)
  {
    // Collaborative loading of M and N tiles into shared memory
    if (Row < numARows && (q * TILE_WIDTH + tx) < numAColumns)
      subTileA[ty][tx] = A[Row * numAColumns + q * TILE_WIDTH + tx];
    else
      subTileA[ty][tx] = 0;
    if (Col < numBColumns && (q * TILE_WIDTH + ty) < numBRows)
      subTileB[tx][ty] = B[b*numBColumns* numBRows + (q * TILE_WIDTH + ty) * numBColumns + Col];
    else
      subTileB[tx][ty] = 0;
    __syncthreads();
    for (int k = 0; k < TILE_WIDTH; ++k)
      Pvalue += subTileA[ty][k] * subTileB[tx][k];
    __syncthreads();
  }
  if (Row < numCRows && Col < numCColumns)
    C[b * numCColumns * numCRows + Row * numCColumns + Col] = Pvalue;
}

__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
  // Allocate memory and copy over the relevant data structures to the GPU

  // We pass double pointers for you to initialize the relevant device pointers,
  //  which are passed to the other two functions.

  // Useful snippet for error checking
  // hipError_t error = hipGetLastError();
  // if(error != hipSuccess)
  // {
  //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
  //     exit(-1);
  // }
  const int H_out = (H - K) / S + 1;
  const int W_out = (W - K) / S + 1;

  hipMalloc(device_output_ptr, B * M * H_out * W_out * sizeof(float));
  hipMalloc(device_input_ptr, B * C * H * W * sizeof(float));
  hipMalloc(device_mask_ptr, M * C * K * K * sizeof(float));

  hipMemcpy(*device_input_ptr, host_input, B * C * H * W * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(*device_mask_ptr, host_mask, M * C * K * K * sizeof(float), hipMemcpyHostToDevice);
}

__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
  // Set the kernel dimensions and call the kernel
  const int H_out = (H - K) / S + 1;
  const int W_out = (W - K) / S + 1;

  int unrolled_W = H_out * W_out;
  int unrolled_H = C * K * K;

  int W_size = (unrolled_W + TILE_WIDTH - 1) / TILE_WIDTH;
  int H_size = (unrolled_H + TILE_WIDTH - 1) / TILE_WIDTH;

  float *device_input_unroll;
  hipMalloc(&device_input_unroll, B * unrolled_H * unrolled_W * sizeof(float));

  dim3 blockDim(TILE_WIDTH, TILE_WIDTH, 1);
  dim3 gridDim(W_size,H_size , B);
  input_matrix_unroll<<<gridDim, blockDim>>>(device_input, device_input_unroll, B, C, H, W, K, S);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
  {
    std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
    exit(-1);
  }

  int numARows = M;
  int numAColumns = unrolled_H;
  int numBRows = unrolled_H;
  int numBColumns = unrolled_W;
  int numCRows = M;
  int numCColumns = unrolled_W;

  dim3 blockDim2(TILE_WIDTH, TILE_WIDTH, 1);

  int W_size2 = (numCColumns + TILE_WIDTH - 1) / TILE_WIDTH;
  int H_size2 = (numCRows + TILE_WIDTH - 1) / TILE_WIDTH;

  dim3 gridDim2(W_size2, H_size2, B);

  matrixMultiplyShared<<<gridDim2, blockDim2>>>(device_mask, device_input_unroll, device_output, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  hipFree(device_input_unroll);
  error = hipGetLastError();
  if (error != hipSuccess)
  {
    std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
    exit(-1);
  }
}

__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
  const int H_out = (H - K) / S + 1;
  const int W_out = (W - K) / S + 1;
  // Copy the output back to host
  hipMemcpy(host_output, device_output, B * M * H_out * W_out * sizeof(float), hipMemcpyDeviceToHost);
  // Free device memory
  hipFree(device_output);
  hipFree(device_input);
  hipFree(device_mask);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess)
  {
    std::cout << "CUDA error: " << hipGetErrorString(error) << std::endl;
    exit(-1);
  }
}

__host__ void GPUInterface::get_device_properties()
{
  int deviceCount;
  hipGetDeviceCount(&deviceCount);

  for (int dev = 0; dev < deviceCount; dev++)
  {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);

    std::cout << "Device " << dev << " name: " << deviceProp.name << std::endl;
    std::cout << "Computational capabilities: " << deviceProp.major << "." << deviceProp.minor << std::endl;
    std::cout << "Max Global memory size: " << deviceProp.totalGlobalMem << std::endl;
    std::cout << "Max Constant memory size: " << deviceProp.totalConstMem << std::endl;
    std::cout << "Max Shared memory size per block: " << deviceProp.sharedMemPerBlock << std::endl;
    std::cout << "Max threads per block: " << deviceProp.maxThreadsPerBlock << std::endl;
    std::cout << "Max block dimensions: " << deviceProp.maxThreadsDim[0] << " x, " << deviceProp.maxThreadsDim[1] << " y, " << deviceProp.maxThreadsDim[2] << " z" << std::endl;
    std::cout << "Max grid dimensions: " << deviceProp.maxGridSize[0] << " x, " << deviceProp.maxGridSize[1] << " y, " << deviceProp.maxGridSize[2] << " z" << std::endl;
    std::cout << "Warp Size: " << deviceProp.warpSize << std::endl;
  }
}
